
#include <hip/hip_runtime.h>
// MIT License

// Copyright (c) 2018 Changan Wang

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#if GOOGLE_CUDA == 1
#define EIGEN_USE_GPU
#include "ps_roi_align_op.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/tensor_shape.h"

using namespace tensorflow;

#include <cstdint>
#include <cmath>
#include <cfloat>

// Define the CUDA kernel.
template <typename T>
__global__ void PSROIAlignGradCudaKernel(CudaLaunchConfig config, const T * inputs, const T * rois, const T * pooled_features_grad, const int32_t * pooled_index, T * grad_output, const int32_t grid_dim_width, const int32_t grid_dim_height, const int batch_size, const int num_channals, const int map_height, const int map_width, const int num_rois) {

  const int32_t grid_size = grid_dim_width * grid_dim_height;
  const int32_t bank_size = num_channals / grid_size;

  CUDA_1D_KERNEL_LOOP(worker_index, config.virtual_thread_count) {
    // image_index * roi_index * channal_pos_remainder * row_index * col_index
    const int32_t position_index = (worker_index % num_channals) / bank_size;
    const int32_t row_index = position_index / grid_dim_width;
    const int32_t col_index = position_index % grid_dim_width;
    // position of the channal of pooled feature
    // position of the channal in the bank of feature map
    const int32_t channal_pos_remainder = worker_index % bank_size;
    const int32_t pool_index = worker_index / num_channals;
    const int32_t image_index = pool_index / num_rois;
    const int32_t roi_index = pool_index % num_rois;

    const T * roi_to_pool = rois + (image_index * num_rois + roi_index) * 4;

    if(ldg(roi_to_pool + 2) < std::numeric_limits<T>::min() || ldg(roi_to_pool + 3) < std::numeric_limits<T>::min()) continue;
    // T roi_ymin = static_cast<T>(0);
    // T roi_xmin = static_cast<T>(0);
    // T roi_ymax = static_cast<T>(0);
    // T roi_xmax = static_cast<T>(0);
    // fix ROI
    // std::tie(roi_ymin, roi_xmin, roi_ymax, roi_xmax) = [roi_to_pool, map_height, map_width](){
    T _roi_y_center = static_cast<T>(ldg(roi_to_pool) * map_height);
    T _roi_x_center = static_cast<T>(ldg(roi_to_pool + 1) * map_width);
    T _roi_h = tf_max(ldg(roi_to_pool + 2) * map_height, static_cast<T>(1));
    T _roi_w = tf_max(ldg(roi_to_pool + 3) * map_width, static_cast<T>(1));

    T roi_ymin = tf_max(_roi_y_center - static_cast<T>(_roi_h / 2.), static_cast<T>(0));
    T roi_xmin = tf_max(_roi_x_center - static_cast<T>(_roi_w / 2.), static_cast<T>(0));
    T roi_ymax = tf_min(_roi_y_center + static_cast<T>(_roi_h / 2.), static_cast<T>(map_height) - std::numeric_limits<T>::min());
    T roi_xmax = tf_min(_roi_x_center + static_cast<T>(_roi_w / 2.), static_cast<T>(map_width) - std::numeric_limits<T>::min());
    //   return std::make_tuple(roi_ymin, roi_xmin, roi_ymax, roi_xmax);
    // }();

    T roi_h = roi_ymax - roi_ymin;
    T roi_w = roi_xmax - roi_xmin;
    float pool_bin_width = static_cast<float>(roi_w) / grid_dim_width;
    float pool_bin_height = static_cast<float>(roi_h) / grid_dim_height;
    int32_t num_elem_width = static_cast<int32_t>(pool_bin_width) + 1;
    int32_t num_elem_height = static_cast<int32_t>(pool_bin_height) + 1;

    // std::cout << "pool_bin_width: " << pool_bin_width << " pool_bin_height: " << pool_bin_height << " num_elem_width: " << num_elem_width << " num_elem_height: " << num_elem_height << std::endl;

    // std::cout << "worker_index: " << worker_index << " roi_index: " << roi_index
    // << " roi_ymin: " << roi_ymin << " roi_xmin: " << roi_xmin << " roi_ymax: " << roi_ymax << " roi_xmax: " << roi_xmax << " image_index: " << image_index << " position_index: " << (position_index % grid_size) << " channal_pos_remainder: " << channal_pos_remainder << std::endl;

    float step_width_each_bin = pool_bin_width / num_elem_width;
    float step_height_each_bin = pool_bin_height / num_elem_height;

    T * grad_output_start = reinterpret_cast<T*>(grad_output + (image_index * num_channals + position_index * bank_size + channal_pos_remainder) * map_height * map_width);

    const T * pooled_features_start = pooled_features_grad + worker_index;
    const int32_t * pooled_index_start = pooled_index + worker_index;
    // T * pooled_features_start = pooled_features_grad + image_index * (num_rois * num_channals) + roi_index * num_channals + (position_index % grid_size) * bank_size + channal_pos_remainder;
    // int32_t * pooled_index_start = pooled_index + image_index * (num_rois * num_channals) + roi_index * num_channals + (position_index % grid_size) * bank_size + channal_pos_remainder;

    float pool_width_start = roi_xmin + pool_bin_width * col_index;
    float pool_height_start = roi_ymin + pool_bin_height * row_index;

    const int32_t h_ind = ldg(pooled_index_start) / num_elem_width;
    const int32_t w_ind = ldg(pooled_index_start) % num_elem_width;

    float col_to_pool = pool_width_start + step_width_each_bin * w_ind + step_width_each_bin / 2.;
    float row_to_pool = pool_height_start + step_height_each_bin * h_ind + step_height_each_bin / 2.;
    //std::cout << "col_to_pool: " << col_to_pool << " row_to_pool: " << row_to_pool << std::endl;
    int32_t int_col_to_pool = static_cast<int32_t>(col_to_pool);
    int32_t int_row_to_pool = static_cast<int32_t>(row_to_pool);
    float float_col_to_pool = col_to_pool - int_col_to_pool;
    float float_row_to_pool = row_to_pool - int_row_to_pool;

    const T grad_in = ldg(pooled_features_start);

    atomicAdd(grad_output_start + int_row_to_pool * map_width + int_col_to_pool, static_cast<T>((1. - float_col_to_pool) * (1. - float_row_to_pool) * grad_in));
    atomicAdd(grad_output_start + tf_min(int_row_to_pool + 1, map_height - 1) * map_width + int_col_to_pool, static_cast<T>((1. - float_col_to_pool) * float_row_to_pool * grad_in));
    atomicAdd(grad_output_start + int_row_to_pool * map_width + tf_min(int_col_to_pool + 1, map_width - 1), static_cast<T>(float_col_to_pool * (1. - float_row_to_pool) * grad_in));
    atomicAdd(grad_output_start + tf_min(int_row_to_pool + 1, map_height - 1) * map_width + tf_min(int_col_to_pool + 1, map_width - 1), static_cast<T>(float_col_to_pool * float_row_to_pool * grad_in));
  }
}

template <typename T>
void PSROIAlignGradFunctor<GPUDevice, T>::operator()(OpKernelContext* context, const GPUDevice& d, typename TTypes<T>::ConstFlat inputs, typename TTypes<T>::ConstFlat rois, const int32_t grid_dim_width, const int32_t grid_dim_height, typename TTypes<T>::ConstFlat pooled_features_grad, typename TTypes<int32_t>::ConstFlat pooled_index, typename TTypes<T>::Flat grad_output, KDimSize dim_info) {

    int batch_size = 0;
    int num_channals = 0;
    int map_height = 0;
    int map_width = 0;
    int num_rois = 0;

    std::tie(batch_size, num_channals, map_height, map_width, num_rois) = dim_info;

    CudaLaunchConfig config = GetCudaLaunchConfig(batch_size * num_rois * num_channals, d);
    //grad_output = grad_output.setZero();
    SetZero <<<config.block_count, config.thread_per_block, 0, d.stream()>>> (batch_size * map_height * map_width * num_channals, grad_output.data());

    PSROIAlignGradCudaKernel <<<config.block_count,
                        config.thread_per_block, 0, d.stream()>>> (config, inputs.data(), rois.data(), pooled_features_grad.data(), pooled_index.data(), grad_output.data(), grid_dim_width, grid_dim_height, batch_size, num_channals, map_height, map_width, num_rois);

    cudaError_t err = cudaGetLastError();
    if(cudaSuccess != err)
    {
      fprintf( stderr, "cudaCheckError() failed : %s\n", cudaGetErrorString( err ) );
      exit( -1 );
    }
}

template struct PSROIAlignGradFunctor<GPUDevice, float>;
// #define DEFINE_GPU_SPECS(T)   \
//   template struct PSROIAlignFunctorGPU<T>;

// TF_CALL_GPU_NUMBER_TYPES(DEFINE_GPU_SPECS);

#endif  // GOOGLE_CUDA
