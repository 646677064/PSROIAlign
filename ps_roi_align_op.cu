
#include <hip/hip_runtime.h>
// MIT License

// Copyright (c) 2018 Changan Wang

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#if GOOGLE_CUDA == 1
#define EIGEN_USE_GPU
#include "ps_roi_align_op.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/tensor_shape.h"

using namespace tensorflow;

#include <cstdint>
#include <cmath>
#include <cfloat>

// Define the CUDA kernel.
template <typename T>
__global__ void PSROIAlignCudaKernel(CudaLaunchConfig config, const T * inputs, const T * rois, T * pooled_features, int32_t * pooled_index, const int32_t grid_dim_width, const int32_t grid_dim_height, const int batch_size, const int num_channals, const int map_height, const int map_width, const int num_rois, const bool using_max_pool) {

  const int32_t grid_size = grid_dim_width * grid_dim_height;
  const int32_t bank_size = num_channals / grid_size;

  CUDA_1D_KERNEL_LOOP(worker_index, config.virtual_thread_count) {
    // image_index * roi_index * channal_pos_remainder * row_index * col_index
    const int32_t position_index = (worker_index % num_channals) / bank_size;
    const int32_t row_index = position_index / grid_dim_width;
    const int32_t col_index = position_index % grid_dim_width;
    // position of the channal of pooled feature
    // position of the channal in the bank of feature map
    const int32_t channal_pos_remainder = worker_index % bank_size;
    const int32_t pool_index = worker_index / num_channals;
    const int32_t image_index = pool_index / num_rois;
    const int32_t roi_index = pool_index % num_rois;

    const T * roi_to_pool = rois + (image_index * num_rois + roi_index) * 4;

    const T * feature_map_to_pool = inputs + (image_index * num_channals + (position_index % grid_size) * bank_size + channal_pos_remainder) * map_height * map_width;
    T * pooled_features_start = pooled_features + image_index * (num_rois * num_channals) + roi_index * num_channals + (position_index % grid_size) * bank_size + channal_pos_remainder;
    int32_t * pooled_index_start = pooled_index + image_index * (num_rois * num_channals) + roi_index * num_channals + (position_index % grid_size) * bank_size + channal_pos_remainder;

    if(roi_to_pool[2] < std::numeric_limits<T>::min() || roi_to_pool[3] < std::numeric_limits<T>::min()){
      *pooled_features_start = static_cast<T>(0);
      continue;
    }
    // T roi_ymin = static_cast<T>(0);
    // T roi_xmin = static_cast<T>(0);
    // T roi_ymax = static_cast<T>(0);
    // T roi_xmax = static_cast<T>(0);
    // fix ROI
    // std::tie(roi_ymin, roi_xmin, roi_ymax, roi_xmax) = [roi_to_pool, map_height, map_width](){
    T _roi_y_center = static_cast<T>(ldg(roi_to_pool) * map_height);
    T _roi_x_center = static_cast<T>(ldg(roi_to_pool + 1) * map_width);
    T _roi_h = tf_max(ldg(roi_to_pool + 2) * map_height, static_cast<T>(1));
    T _roi_w = tf_max(ldg(roi_to_pool + 3) * map_width, static_cast<T>(1));

    T roi_ymin = tf_max(_roi_y_center - static_cast<T>(_roi_h / 2.), static_cast<T>(0));
    T roi_xmin = tf_max(_roi_x_center - static_cast<T>(_roi_w / 2.), static_cast<T>(0));
    T roi_ymax = tf_min(_roi_y_center + static_cast<T>(_roi_h / 2.), static_cast<T>(map_height) - std::numeric_limits<T>::min());
    T roi_xmax = tf_min(_roi_x_center + static_cast<T>(_roi_w / 2.), static_cast<T>(map_width) - std::numeric_limits<T>::min());
    //   return std::make_tuple(roi_ymin, roi_xmin, roi_ymax, roi_xmax);
    // }();

    T roi_h = roi_ymax - roi_ymin;
    T roi_w = roi_xmax - roi_xmin;
    float pool_bin_width = static_cast<float>(roi_w) / grid_dim_width;
    float pool_bin_height = static_cast<float>(roi_h) / grid_dim_height;
    int32_t num_elem_width = static_cast<int32_t>(pool_bin_width) + 1;
    int32_t num_elem_height = static_cast<int32_t>(pool_bin_height) + 1;

    // std::cout << "pool_bin_width: " << pool_bin_width << " pool_bin_height: " << pool_bin_height << " num_elem_width: " << num_elem_width << " num_elem_height: " << num_elem_height << std::endl;

    // std::cout << "worker_index: " << worker_index << " roi_index: " << roi_index
    // << " roi_ymin: " << roi_ymin << " roi_xmin: " << roi_xmin << " roi_ymax: " << roi_ymax << " roi_xmax: " << roi_xmax << " image_index: " << image_index << " position_index: " << (position_index % grid_size) << " channal_pos_remainder: " << channal_pos_remainder << std::endl;

    float step_widht_each_bin = pool_bin_width / num_elem_width;
    float step_height_each_bin = pool_bin_height / num_elem_height;

    float pool_width_start = roi_xmin + pool_bin_width * col_index;
    float pool_height_start = roi_ymin + pool_bin_height * row_index;
    int32_t max_pool_ind = 0;
    //T max_elem = std::numeric_limits<T>::lowest();
    T max_or_acc_elem = using_max_pool ? std::numeric_limits<T>::lowest() : static_cast<T>(0);
    for (int32_t h_ind = 0; h_ind < num_elem_height; ++h_ind) {
      for (int32_t w_ind = 0; w_ind < num_elem_width; ++w_ind) {
        float col_to_pool = pool_width_start + step_widht_each_bin * w_ind + step_widht_each_bin / 2.;
        float row_to_pool = pool_height_start + step_height_each_bin * h_ind + step_height_each_bin / 2.;
        //std::cout << "col_to_pool: " << col_to_pool << " row_to_pool: " << row_to_pool << std::endl;
        int32_t int_col_to_pool = static_cast<int32_t>(col_to_pool);
        int32_t int_row_to_pool = static_cast<int32_t>(row_to_pool);
        float float_col_to_pool = col_to_pool - int_col_to_pool;
        float float_row_to_pool = row_to_pool - int_row_to_pool;

        int32_t current_switch_ind = num_elem_width * h_ind + w_ind;
        //std::cout << "current_switch_ind: " << current_switch_ind << std::endl;
        T temp_value = static_cast<T>((1. - float_col_to_pool) * (1. - float_row_to_pool) * ldg(feature_map_to_pool + int_row_to_pool * map_width + int_col_to_pool) +
                                  (1. - float_col_to_pool) * float_row_to_pool * ldg(feature_map_to_pool + tf_min(int_row_to_pool + 1, map_height - 1) * map_width + int_col_to_pool) +
                                  float_col_to_pool * (1. - float_row_to_pool) * ldg(feature_map_to_pool + int_row_to_pool * map_width + tf_min(int_col_to_pool + 1, map_width - 1)) +
                                  float_col_to_pool * float_row_to_pool * ldg(feature_map_to_pool + tf_min(int_row_to_pool + 1, map_height - 1) * map_width + tf_min(int_col_to_pool + 1, map_width - 1)));
        if(using_max_pool){
          if(max_or_acc_elem < temp_value){
            max_or_acc_elem = temp_value;
            max_pool_ind = current_switch_ind;
          }
        }else{
          max_or_acc_elem += temp_value;
        }
      }
    }
    if(!using_max_pool) max_or_acc_elem /= static_cast<T>(num_elem_height * num_elem_width);
    *pooled_features_start = max_or_acc_elem;
    *pooled_index_start = using_max_pool ? max_pool_ind : static_cast<T>(0);
  }
}

template <typename T>
void PSROIAlignFunctor<GPUDevice, T>::operator()(OpKernelContext* context, const GPUDevice& d, typename TTypes<T>::ConstFlat inputs, typename TTypes<T>::ConstFlat rois, const int32_t grid_dim_width, const int32_t grid_dim_height, typename TTypes<T>::Flat pooled_features, typename TTypes<int32_t>::Flat pooled_index, KDimSize dim_info) {

    int batch_size = 0;
    int num_channals = 0;
    int map_height = 0;
    int map_width = 0;
    int num_rois = 0;
    bool using_max_pool = false;

    std::tie(batch_size, num_channals, map_height, map_width, num_rois, using_max_pool) = dim_info;

    CudaLaunchConfig config = GetCudaLaunchConfig(batch_size * num_rois * num_channals, d);
    PSROIAlignCudaKernel <<<config.block_count,
                        config.thread_per_block, 0, d.stream()>>> (config, inputs.data(), rois.data(), pooled_features.data(), pooled_index.data(), grid_dim_width, grid_dim_height, batch_size, num_channals, map_height, map_width, num_rois, using_max_pool);

    cudaError_t err = cudaGetLastError();
    if(cudaSuccess != err)
    {
      fprintf( stderr, "cudaCheckError() failed : %s\n", cudaGetErrorString( err ) );
      exit( -1 );
    }
}

template struct PSROIAlignFunctor<GPUDevice, float>;
// #define DEFINE_GPU_SPECS(T)   \
//   template struct PSROIAlignFunctorGPU<T>;

// TF_CALL_GPU_NUMBER_TYPES(DEFINE_GPU_SPECS);

#endif  // GOOGLE_CUDA
